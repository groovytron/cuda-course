#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(int* tabSM, int* ptrTabIn, uint lenght);

__device__ void reductionInterBlock(int* tabSM, int* ptrHistogrammeGM);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Constructeur		*|
 \*-------------------------------------*/

__global__ void histogramme(int* ptrTabIn, uint lenght, int* ptrHistogrammeGM)
    {
    extern __shared__ int tabSM[];

    reductionIntraThread(tabSM, ptrTabIn, lenght);

    __syncthreads();

    reductionInterBlock(tabSM, ptrHistogrammeGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(int* tabSM, int* ptrTabIn, uint lenght)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    int s = TID;

    while (s < lenght)
	{
	int value = ptrTabIn[s];
	atomicAdd(&tabSM[value], 1);
	s += NB_THREAD;
	}
    }

__device__ void reductionInterBlock(int* tabSM, int* ptrHistogrammeGM)
    {
    if (Indice2D::tidLocal() == 0)
	{
	for (int i = 0; i < 256; i++)
	    {
	    atomicAdd(&ptrHistogrammeGM[i], tabSM[i]);
	    }
	}
    }
