#include "hip/hip_runtime.h"
#include <iostream>
#include "Device.h"
#include "cudaTools.h"
#include "DataCreator.h"

#include "Histogramme.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void histogramme(int* ptrTabInput, uint length, int* ptrHistogrammeGM);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Constructeur		*|
 \*-------------------------------------*/

Histogramme::Histogramme(const Grid& grid, int* ptrResult, int dataMax)
    {

    this->dg = grid.dg;
    this->db = grid.db;

    this->ptrResult = ptrResult;

    DataCreator dataCreator(dataMax);
    this->data = dataCreator.getTabData();
    this->length = dataCreator.getLength();

    // Memory management
    this->sizeOctetResult = sizeof(int) * dataMax;
    this->sizeOctetData = sizeof(int) * this->length;

    Device::malloc(&ptrTabInput, sizeOctetData);
    Device::memclear(ptrTabInput, sizeOctetData);
    Device::memcpyHToD(ptrTabInput, data, sizeOctetData);

    Device::malloc(&ptrTabOutput, sizeOctetResult);
    Device::memclear(ptrTabOutput, sizeOctetResult);

    }

Histogramme::~Histogramme(void)
    {
    Device::free(ptrTabInput);
    Device::free(ptrTabOutput);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Histogramme::run()
    {
    Device::lastCudaError("histogramme (before)");
    histogramme<<<dg,db, sizeOctetResult>>>(ptrTabInput, length, ptrTabOutput);
    Device::lastCudaError("histogramme (after)");

    Device::synchronize();

    Device::memcpyDToH(ptrResult, ptrTabOutput, sizeOctetResult);
    }

