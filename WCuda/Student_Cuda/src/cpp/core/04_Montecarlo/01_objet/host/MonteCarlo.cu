#include "hip/hip_runtime.h"
#include <iostream>
#include "Device.h"
#include "cudaTools.h"
#include "MonteCarlo.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

// Kernel 1
extern __global__ void setup_kernel_rand(hiprandState* tabDevGeneratorGM, int deviceId);
// Kernel 2
extern __global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbFleches,uint m, uint* ptrDevNxTotal);

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

MonteCarlo::MonteCarlo(const Grid& grid, int nbFlechesTotal, float m)
    {

    this->dg = grid.dg;
    this->db = grid.db;

    this->nbThread = Device::nbThread(grid);
    this->nbFlechesTotal = nbFlechesTotal;
    this->m = m;
    this->nbFleches = nbFlechesTotal / nbThread;

    this->ptrDevGeneratorGM = NULL;
    this->sizeOctetGeneratorGM = nbThread * sizeof(hiprandState);

    this->result = 0;
    this->ptrDevResult = NULL;
    this->sizeInt = sizeof(uint);

    this->sizeTabSM = db.x * sizeof(uint);

    Device::malloc(&ptrDevGeneratorGM, sizeOctetGeneratorGM);
    Device::memclear(ptrDevGeneratorGM, sizeOctetGeneratorGM);

    int id = Device::getDeviceId();

    setup_kernel_rand<<<dg,db>>>(ptrDevGeneratorGM,id);

    Device::malloc(&ptrDevResult, sizeInt);
    hipMemset(ptrDevResult, 0, sizeInt);
    }

MonteCarlo::~MonteCarlo(void)
    {
    Device::free(ptrDevGeneratorGM);
    Device::free(ptrDevResult);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void MonteCarlo::run()
    {

    montecarlo<<<dg,db, sizeTabSM>>>(ptrDevGeneratorGM, nbFleches,m, ptrDevResult); // assynchrone

    Device::memcpyDToH(&montecarloResult, ptrDevResult, sizeInt); // barriere synchronisation implicite

    result = (double) montecarloResult / (double) nbFlechesTotal * m;
    }

float MonteCarlo::getResult()
    {
    return result;
    }

uint MonteCarlo::getMontecarloResult()
    {
    return montecarloResult;
    }
