#include "hip/hip_runtime.h"
#include "LimitsTools.h"
#include "Indice1D.h"
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel_rand(hiprandState* tabDevGeneratorGM, int deviceId)
    {
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGeneratorGM[TID]);
    }
