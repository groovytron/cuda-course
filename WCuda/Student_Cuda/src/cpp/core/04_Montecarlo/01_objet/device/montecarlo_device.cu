#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "reductionADD.h"
#include <hiprand/hiprand_kernel.h>

__global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbALancer, uint m, uint* ptrDevNxTotal);

__device__ float f(float x);

static __device__ void reductionIntraThread(hiprandState* tabDevGeneratorGM, uint nbALancer, uint m, uint* ptrDevNxTotal, uint* tabSM);

__global__ void montecarlo(hiprandState* tabDevGeneratorGM, uint nbALancer, uint m, uint* ptrDevNxTotal)
    {
    extern __shared__ uint
    tabSM[];

    reductionIntraThread(tabDevGeneratorGM, nbALancer, m, ptrDevNxTotal, tabSM);

    __syncthreads();

    reductionADD < uint > (tabSM, ptrDevNxTotal);
    }

static __device__ void reductionIntraThread(hiprandState* tabDevGeneratorGM, uint nbALancer, uint m, uint* ptrDevNxTotal, uint* tabSM)
    {
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    hiprandState localGenerator = tabDevGeneratorGM[TID];

    int nx = 0;
    float x;
    float y;

    for(int i = 0; i < nbALancer; i++)
	{

	x = hiprand_uniform(&localGenerator);
	y = hiprand_uniform(&localGenerator) * m;

	if (y < f(x))
	    {
	    nx ++;
	    }
	}

    tabSM[TID_LOCAL] = nx;
    tabDevGeneratorGM[TID_LOCAL] = localGenerator;
    }

__device__ float f(float x)
    {
    return 4 / (1 + x * x);
    }
