#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

// Kernel 1
__global__
void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

// Kernel 2
__global__
void montecarlo(hiprandState* ptrTabDevGenerator, int nbFlecheByThread, int* ptrDevNxTotal);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid, int nbFlecheByThread) :
	nbFlecheByThread(nbFlecheByThread)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    // TODO: sizeOcteSM, sizeOctetGenerator, sizeOctetResultatFinal
    this->sizeOctetGMResult = sizeof(int);
    this->sizeOctetGMGenerator = sizeof(hiprandState) * grid.threadCounts();
    this->sizeOctetSM = sizeof(int) * this->db.x; // octet
    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevGMResult, sizeOctetGMResult);
	    Device::memclear(ptrDevGMResult, sizeOctetGMResult);
	    Device::malloc(&ptrDevGMGenerator, sizeOctetGMGenerator);
	    Device::memclear(ptrDevGMGenerator, sizeOctetGMGenerator);
	    }

	Device::lastCudaError("Montecarlo MM (end allocation)"); // temp debug, facultatif
	}

    // Kernel 1. fait ici pour optimisation
setup_kernel_rand<<<this->dg, this->db>>>(ptrDevGMGenerator, Device::getDeviceId());
}

Montecarlo::~Montecarlo(void)
{
//MM (device free)
    {
    Device::free(ptrDevGMResult);
    Device::free(ptrDevGMGenerator);
    Device::lastCudaError("Montecarlo MM (end deallocation)"); // temp debug, facultatif
    }
}

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Montecarlo::run()
{
Device::lastCudaError("Montecarlo (before)"); // temp debug
montecarlo<<<dg, db, sizeOctetSM>>>(ptrDevGMGenerator, nbFlecheByThread, ptrDevGMResult); // assynchrone

Device::lastCudaError("Montecarlo (after)");

int result = 0;

// MM (Device -> Host)
    {
    Device::memcpyDToH(&result, ptrDevGMResult, sizeOctetGMResult); // barriere synchronisation implicite
    }

piAt = result;
//	piAt = (result / (float)nbFlecheByThread) * 4.f;
}

int Montecarlo::getResult()
{
return piAt;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
