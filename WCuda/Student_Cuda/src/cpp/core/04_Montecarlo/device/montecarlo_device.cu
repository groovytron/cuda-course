#include "hip/hip_runtime.h"
#include <Indice1D.h>
#include <Indice2D.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

// Kernel 2
__global__ void montecarlo(hiprandState* ptrTabDevGenerator, int nbFlecheByThread, int* ptrDevNxTotal); // ptrDevNxTotal est en GM

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(hiprandState* tabDevGenerator, int n, int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__
void montecarlo(hiprandState* ptrTabDevGenerator, int nbFlecheByThread, int* ptrDevNxTotal)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread(ptrTabDevGenerator, nbFlecheByThread, tabSM);
    __syncthreads();
    reductionADD<int>(tabSM, ptrDevNxTotal);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(hiprandState* tabDevGenerator, int n, int* tabSM)
    {
    const int TID = Indice1D::tidLocal();

    // Global Memory -> Register (optimization)
    hiprandState localGenerator = tabDevGenerator[TID];

//    float xAlea;
//    float yAlea;
//    int nX = 0;

//    for (int i = 1; i <= n; i++)
//	{
//	xAlea = hiprand_uniform(&localGenerator);
//	yAlea = hiprand_uniform(&localGenerator) * 4.f;
//
//	if (y <= f(xAlea))
//	    {
//		// Flèche dans la cible
//	    nX += 1;
//	    }
//	else
//	    {
//		// FLèche hors de la cible
//	    }

	// ...
	// work(xAlea, yAlea); LE CALCUL PEUT SE FAIRE SANS FONCTION
	// ...
//	}

    // Register -> Global Memory
    // Necessaire si on veut utiliser notre generator
    // -dans d’ autre kernel
    // - avec d’ autres nombres aleatoires!

    tabSM[TID] = 1;
    tabDevGenerator[TID] = localGenerator;
    }

__device__ float f(float x)
    {
    return 1 / (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

