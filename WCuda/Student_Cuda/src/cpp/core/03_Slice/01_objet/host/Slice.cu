#include "hip/hip_runtime.h"
#include "Slice.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

//extern __global__ void slice(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);
extern __global__ void slice(float* ptrDevGMResult, int nbSlice);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int nbSlice) :
	nbSlice(nbSlice)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}

    this->sizeOctetGM = sizeof(float);
    this->sizeOctetSM = sizeof(float) * this->db.x; // octet
    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrDevGMResult, sizeOctetGM);
	    Device::memclear(ptrDevGMResult, sizeOctetGM);
	    }

	Device::lastCudaError("Slice MM (end allocation)"); // temp debug, facultatif
	}

    }

Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevGMResult);
	Device::lastCudaError("Slice MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    Device::lastCudaError("slice (before)"); // temp debug
    slice<<<dg, db, sizeOctetSM>>>(ptrDevGMResult, nbSlice); // assynchrone
    Device::lastCudaError("slice (after)");

    // MM (Device -> Host)
	{
	Device::memcpyDToH(&result, ptrDevGMResult, sizeOctetGM); // barriere synchronisation implicite
	}

    }

float Slice::getResult()
    {
    return result;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
