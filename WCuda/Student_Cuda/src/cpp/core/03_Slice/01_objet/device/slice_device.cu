#include "hip/hip_runtime.h"
#include <Indice1D.h>
#include <Indice2D.h>
#include <stdio.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void slice(float* ptrDevGMResult, int nbSlice);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float fpi(float x);
static __device__ void reductionIntraThread(float* tabSM, int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__
void slice(float* ptrDevGMResult, int nbSlice)
    {
    extern __shared__ float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();
    reductionADD<float>(tabSM, ptrDevGMResult);
//    reductionAdd<float>(tabSM, ptrDevGMResult);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(float* tabSM, int nbSlice)
    {
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();

    const int TID_LOCAL = Indice1D::tidLocal();
    const float DX = 1.0f / (float) nbSlice;
    float sommeLocale = 0;

    int s = TID;
    while (s < nbSlice)
	{
	float x = s * DX;
	sommeLocale += fpi(x);
	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = 4 *sommeLocale * DX;
    }

__device__ float fpi(float x)
    {
    return 1 / (1. + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

