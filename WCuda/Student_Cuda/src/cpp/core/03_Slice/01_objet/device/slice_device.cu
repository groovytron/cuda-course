#include "hip/hip_runtime.h"
#include <Indice1D.h>
#include <Indice2D.h>
#include <stdio.h>
#include "reductionADD.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void slice(float* ptrDevGMResult, int nbSlice);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float aire(int s);
static __device__ float fpi(int s);
static __device__ void reductionIntraThread(float* tabSM, int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__
void slice(float* ptrDevGMResult, int nbSlice)
    {
    extern __shared__ float tabSM[];
    reductionIntraThread(tabSM, nbSlice);
    __syncthreads();
    reductionADD<float>(tabSM, ptrDevGMResult);
//    reductionAdd<float>(tabSM, ptrDevGMResult);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reductionIntraThread(float* tabSM, int nbSlice)
    {
    const int NB_THREAD = Indice1D::nbThread();
    const int TID = Indice1D::tid();

    const int TID_LOCAL = Indice1D::tidLocal();
//    const float DX = 1.0f / (float) nbSlice; TODO: use for final result computation
    float sommeLocale = 0;

    int s = TID;
    while (s < nbSlice)
	{
	sommeLocale += aire(s);
	s += NB_THREAD;
	}

//    tabSM[TID_LOCAL] = s * DX;
    tabSM[TID_LOCAL] = 1.0f;

    }

__device__ float aire(int s)
    {
// TODO: implement
    return fpi(s);
    }

__device__ float fpi(int s)
    {
// TODO: implement
    return 1.0f;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

