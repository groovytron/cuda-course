#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>
#include "RayTracing.h"

using std::cout;
using std::endl;

extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h,float t,uint nbSphere, Sphere* ptrDevTabSphere);

RayTracing::RayTracing(uint nbSphere, const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
    {

    // Inputs
    this->dt = dt;
    this->nbSphere = nbSphere;
    this->sizeOctet = nbSphere * sizeof(Sphere);

    // Tools
    this->t = 0;
    this->dg = grid.dg;
    this->db = grid.db;

    SphereCreator sphereCreator(nbSphere, w, h, RayTracing::BORD);
    Sphere* ptrTabSpheres = sphereCreator.getTabSphere();

    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSpheres, sizeOctet);
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
rayTracing<<<dg,db>>>(ptrDevPixels,w,h,t,nbSphere,ptrDevTabSphere);
//Device::synchronize();

}

void RayTracing::animationStep()
{
t += dt;
}

