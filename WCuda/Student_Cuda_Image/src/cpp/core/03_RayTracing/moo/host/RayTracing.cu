#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>
#include "RayTracing.h"
#include "Sphere.h"
#include "SphereCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(Sphere* ptrDevSpheres, uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt, int nbSpheres) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda")
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;
    this->sizeOctet = nbSpheres * sizeof(Sphere);

    // Tools
    this->t = 0; // protected dans Animable

    this->nbSpheres = nbSpheres;
    const int BORD = 200;
    SphereCreator sphereCreator(nbSpheres, w, h, BORD);
    this->ptrSpheres = sphereCreator.getTabSphere();

    printf("Coucou from Host rayon: %f\n", ptrSpheres[0].getRayon());

    Device::malloc(&ptrDevSpheres, sizeOctet);
    Device::memclear(ptrDevSpheres, sizeOctet);
    Device::memcpyHToD(ptrDevSpheres, ptrSpheres, sizeOctet);

    }

RayTracing::~RayTracing()
    {
    // rien
    Device::free(ptrDevSpheres);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    raytracing <<<db, dg>>>(this->ptrDevSpheres, ptrDevPixels, w, h, t, this->nbSpheres);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

